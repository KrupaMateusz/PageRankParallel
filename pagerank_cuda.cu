
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <unordered_map>
#include <cmath>
#include <algorithm>
#include <chrono>

using namespace std;

#define THREADS_PER_BLOCK 256
#define DAMPING 0.85
#define EPSILON 1e-6
#define MAX_ITER 100

__global__ void pagerank_kernel(
    int N,
    const int* row_ptr,
    const int* col_idx,
    const int* out_degree,
    const double* rank,
    double* new_rank)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    double sum = 0.0;
    for (int j = row_ptr[i]; j < row_ptr[i + 1]; ++j) {
        int src = col_idx[j];
        sum += rank[src] / out_degree[src];
    }
    new_rank[i] = (1.0 - DAMPING) / N + DAMPING * sum;
}

void load_graph_csr(
    const string& filename,
    vector<int>& row_ptr,
    vector<int>& col_idx,
    vector<int>& out_degree,
    int& N)
{
    unordered_map<int, vector<int>> incoming_links;
    unordered_map<int, int> out_deg;

    ifstream infile(filename);
    string line;
    int max_node = 0;

    while (getline(infile, line)) {
        istringstream iss(line);
        int from, to;
        if (!(iss >> from >> to)) continue;
        incoming_links[to].push_back(from);
        out_deg[from]++;
        max_node = max({max_node, from, to});
    }

    N = max_node + 1;
    row_ptr.resize(N + 1, 0);
    out_degree.resize(N, 0);

    for (const auto& [node, links] : incoming_links) {
        row_ptr[node + 1] = links.size();
    }

    for (int i = 1; i <= N; ++i) {
        row_ptr[i] += row_ptr[i - 1];
    }

    col_idx.resize(row_ptr[N]);
    vector<int> offset(N, 0);

    for (const auto& [to, froms] : incoming_links) {
        int pos = row_ptr[to];
        for (int src : froms) {
            col_idx[pos + offset[to]] = src;
            offset[to]++;
        }
    }

    for (const auto& [node, deg] : out_deg) {
        out_degree[node] = deg;
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Użycie: " << argv[0] << " <plik edges.txt>\n";
        return 1;
    }

    vector<int> row_ptr, col_idx, out_degree;
    int N;
    load_graph_csr(argv[1], row_ptr, col_idx, out_degree, N);

    vector<double> rank(N, 1.0 / N), new_rank(N, 0.0);

    // Alokacja GPU
    int *d_row_ptr, *d_col_idx, *d_out_degree;
    double *d_rank, *d_new_rank;

    hipMalloc(&d_row_ptr, sizeof(int) * row_ptr.size());
    hipMalloc(&d_col_idx, sizeof(int) * col_idx.size());
    hipMalloc(&d_out_degree, sizeof(int) * out_degree.size());
    hipMalloc(&d_rank, sizeof(double) * N);
    hipMalloc(&d_new_rank, sizeof(double) * N);

    // Kopiowanie do GPU
    hipMemcpy(d_row_ptr, row_ptr.data(), sizeof(int) * row_ptr.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx.data(), sizeof(int) * col_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_out_degree, out_degree.data(), sizeof(int) * out_degree.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_rank, rank.data(), sizeof(double) * N, hipMemcpyHostToDevice);

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    auto start = chrono::high_resolution_clock::now();

    for (int iter = 0; iter < MAX_ITER; ++iter) {
        pagerank_kernel<<<blocks, THREADS_PER_BLOCK>>>(
            N, d_row_ptr, d_col_idx, d_out_degree, d_rank, d_new_rank
        );

        swap(d_rank, d_new_rank);
    }

    hipMemcpy(rank.data(), d_rank, sizeof(double) * N, hipMemcpyDeviceToHost);

    auto end = chrono::high_resolution_clock::now();
    double time = chrono::duration<double>(end - start).count();

    // Wypisanie 3 największych i 3 najmniejszych
    vector<pair<int, double>> ranked;
    for (int i = 0; i < N; ++i) ranked.emplace_back(i, rank[i]);

    sort(ranked.begin(), ranked.end(),
              [](auto& a, auto& b) { return a.second > b.second; });

    cout << "🕒 Czas działania CUDA PageRank: " << time << " sekund\n\n";
    cout << "n10 największych współczynników PageRank:\n";
    for (int i = 0; i < 10; ++i)
        cout << "Wierzchołek " << ranked[i].first << ": " << ranked[i].second << "\n";

    cout << "\n10 najmniejszych współczynników PageRank:\n";
    for (int i = ranked.size() - 10; i < ranked.size(); ++i)
        cout << "Wierzchołek " << ranked[i].first << ": " << ranked[i].second << "\n";

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_out_degree);
    hipFree(d_rank);
    hipFree(d_new_rank);

    return 0;
}
